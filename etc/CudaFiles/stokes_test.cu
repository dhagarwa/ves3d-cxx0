#include <fstream>
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

extern void cuda_stokes(int, int, float*, float*, float*, float*);

void readfile(float *arr, char *fname, int size) {
  std::ifstream file(fname);
  if (file.is_open()) {
    int idx=0;
    while (idx < size) {
      file >> arr[idx++];
      fprintf(stdout, "%E\n", arr[idx - 1]);
    }
    fprintf(stdout, "Done\n");
    file.close();
  }
}


int main() {
  float *trg, *src, *den, *u, *q;
  int n = 1;
  int m = 544;
  int dim = 3;
  trg = (float*) malloc(sizeof(float) * dim * n);
  u = (float*) malloc(sizeof(float) * dim * n);
  src = (float*) malloc(sizeof(float) * dim * m * n);
  den = (float*) malloc(sizeof(float) * dim * m * n);
  readfile(trg, "targ_p12.txt", dim * n);
  readfile(src, "source_p12.txt", dim * m * n);
  readfile(den, "density_p12.txt", dim * m * n);

  hipSetDevice(0);
  float *trg_dev, *src_dev, *den_dev, *u_dev, *q_dev;
  hipMalloc((void**)&trg_dev, sizeof(float) * dim * n);
    hipError_t C_E = hipGetLastError ();
  fprintf (stderr, "%s\n", hipGetErrorString (C_E));
  hipMalloc((void**)&u_dev, sizeof(float) * dim * n);
  hipMalloc((void**)&src_dev, sizeof(float) * dim * m * n);
  hipMalloc((void**)&q_dev, sizeof(float) * dim * m * n);
  hipMalloc((void**)&den_dev, sizeof(float) * dim * m * n);
  hipMemcpy(trg_dev, trg, sizeof(float) * dim * n, hipMemcpyHostToDevice);
    C_E = hipGetLastError ();
  fprintf (stderr, "%s\n", hipGetErrorString (C_E));
  hipMemcpy(den_dev, den, sizeof(float) * dim * m * n, hipMemcpyHostToDevice);
    C_E = hipGetLastError ();
  fprintf (stderr, "%s\n", hipGetErrorString (C_E));
  hipMemcpy(src_dev, src, sizeof(float) * dim * m * n, hipMemcpyHostToDevice);
    C_E = hipGetLastError ();
  fprintf (stderr, "%s\n", hipGetErrorString (C_E));

  cuda_stokes(m, n, trg_dev, src_dev, den_dev, u_dev);

  hipMemcpy(u, u_dev, sizeof(float) * dim * n, hipMemcpyDeviceToHost);

  for (int i = 0; i < dim * n; i++) {
    fprintf(stderr, "%E\n", u[i]);
  }
  return 0;
}
